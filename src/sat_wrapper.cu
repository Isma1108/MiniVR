#include "hip/hip_runtime.h"
#include "sat_wrapper.cuh"
#include <hip/hip_runtime.h>
#include <vector>
#include <cfloat>

// Global pointers for GPU memory
cudaVector3D* d_triangles = nullptr;

__device__ void projectDevice(const cudaVector3D* points, int numPoints, const cudaVector3D& axis, float& min, float& max) {
    min = FLT_MAX;
    max = -FLT_MAX;
    for (int i = 0; i < numPoints; ++i) {
        float val = points[i].dot(axis);
        if (val < min) min = val;
        if (val > max) max = val;
    }
}

__global__ void intersectsKernel(const BoundingBox box, const int* triangleIndices, int numTriangles, bool* results, const cudaVector3D* d_triangles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numTriangles) return;

    int triIdx = triangleIndices[idx];
    const cudaVector3D* tri = &d_triangles[triIdx * 3];
    cudaVector3D boxVertices[8];

    // Compute the 8 vertices of the bounding box
    for (int i = 0; i < 8; ++i) {
        boxVertices[i] = cudaVector3D(box.min.x, box.min.y, box.min.z);
        if (i & 1) boxVertices[i].x = box.max.x;
        if (i & 2) boxVertices[i].y = box.max.y;
        if (i & 4) boxVertices[i].z = box.max.z;
    }

    // Check box axes
    cudaVector3D boxNormals[3] = {
        cudaVector3D(1, 0, 0),
        cudaVector3D(0, 1, 0),
        cudaVector3D(0, 0, 1)
    };

    for (int i = 0; i < 3; ++i) {
        float boxMin, boxMax;
        projectDevice(boxVertices, 8, boxNormals[i], boxMin, boxMax);

        float triangleMin, triangleMax;
        projectDevice(tri, 3, boxNormals[i], triangleMin, triangleMax);
        if (triangleMax < boxMin || triangleMin > boxMax) {
            results[idx] = false;
            return;
        }
    }

    // Check triangle's normal
    cudaVector3D triangleEdges[3] = {
        tri[1] - tri[0],
        tri[2] - tri[1],
        tri[0] - tri[2]
    };
    cudaVector3D triangleNorm = triangleEdges[0].cross(triangleEdges[1]);
    float triangleOffset = triangleNorm.dot(tri[0]);

    float boxMin, boxMax;
    projectDevice(boxVertices, 8, triangleNorm, boxMin, boxMax);
    if (boxMax < triangleOffset || boxMin > triangleOffset) {
        results[idx] = false;
        return;
    }

    // Check cross-products of triangle edges and box normals
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            cudaVector3D axis = triangleEdges[i].cross(boxNormals[j]);
            projectDevice(boxVertices, 8, axis, boxMin, boxMax);

            float triangleMin, triangleMax;
            projectDevice(tri, 3, axis, triangleMin, triangleMax);
            if (boxMax < triangleMin || boxMin > triangleMax) {
                results[idx] = false;
                return;
            }
        }
    }

    // If no separating axis is found
    results[idx] = true;
}


// After some errors, I've seen that we need to call kernels or runtime CUDA functions via wrappers in the .cu
// files. Doing this, all can be compiled without errors.

namespace Wrapper {
    void loadTrianglesToGPU(const std::vector<Triangle>& h_triangles) {
        size_t triangleSize = h_triangles.size() * 3 * sizeof(cudaVector3D);
        hipMalloc(&d_triangles, triangleSize);

        std::vector<cudaVector3D> h_cudaTriangles;
        h_cudaTriangles.reserve(h_triangles.size() * 3);

        for (const auto& tri : h_triangles) {
            h_cudaTriangles.emplace_back(tri.vertices[0].x, tri.vertices[0].y, tri.vertices[0].z);
            h_cudaTriangles.emplace_back(tri.vertices[1].x, tri.vertices[1].y, tri.vertices[1].z);
            h_cudaTriangles.emplace_back(tri.vertices[2].x, tri.vertices[2].y, tri.vertices[2].z);
        }

        hipMemcpy(d_triangles, h_cudaTriangles.data(), triangleSize, hipMemcpyHostToDevice);
    }

    void freeGPUMemory() {
        hipFree(d_triangles);
    }

    void callIntersectsKernel(const BoundingBox& box, const std::vector<int>& triangleIndices, bool*& results) {
        int numTriangles = triangleIndices.size();

        int* d_triangleIndices = nullptr;
        bool* d_results = nullptr;
        size_t indexSize = numTriangles * sizeof(int);
        size_t resultSize = numTriangles * sizeof(bool);

        hipMalloc(&d_triangleIndices, indexSize);
        hipMalloc(&d_results, resultSize);

        hipMemcpy(d_triangleIndices, triangleIndices.data(), indexSize, hipMemcpyHostToDevice);

        int blockSize = 256;
        int gridSize = (numTriangles + blockSize - 1) / blockSize;
        intersectsKernel<<<gridSize, blockSize>>>(box, d_triangleIndices, numTriangles, d_results, d_triangles);

        results = new bool[numTriangles];
        hipMemcpy(results, d_results, resultSize, hipMemcpyDeviceToHost);

        hipFree(d_triangleIndices);
        hipFree(d_results);
    }
}

